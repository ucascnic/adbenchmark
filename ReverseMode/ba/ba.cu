#include "hip/hip_runtime.h"

#include<hip/hip_runtime.h>
#include"../adbench/header.h" 

__global__ void show_res_(int n, double *data){
    int tid = threadIdx.x;

    printf("ind = %d, data=%.2f\n",tid,data[tid]);
}
void show_res(int n, double *data){
    
    show_res_<<<1,3>>>(n,data);
}

#include <stdio.h>

void __device__ dcompute_reproj_error_cu( double const*, double*, double const*, double*, double const*, double*, double const*, double*, double*);
 
 
int __device__ enzyme_dup;
int __device__ enzyme_const;
int __device__ enzyme_out;
int __device__ enzyme_dupnoneed;

double __device__ sqsum_cu(int n, double const* x)
{
    int i;
    double res = 0;
    for (i = 0; i < n; i++)
    {
        res = res + x[i] * x[i];
    }

    return res;
}



void __device__ cross_cu(double const* a, double const* b, double* out)
{
    out[0] = a[1] * b[2] - a[2] * b[1];
    out[1] = a[2] * b[0] - a[0] * b[2];
    out[2] = a[0] * b[1] - a[1] * b[0];
}



/* ===================================================================== */
/*                               MAIN LOGIC                              */
/* ===================================================================== */

// rot: 3 rotation parameters
// pt: 3 point to be rotated
// rotatedPt: 3 rotated point
// this is an efficient evaluation (part of
// the Ceres implementation)
// easy to understand calculation in matlab:
//  theta = sqrt(sum(w. ^ 2));
//  n = w / theta;
//  n_x = au_cross_matrix(n);
//  R = eye(3) + n_x*sin(theta) + n_x*n_x*(1 - cos(theta));
void __device__ rodrigues_rotate_point_cu(double const* __restrict rot, double const* __restrict pt, double *__restrict rotatedPt)
{
    int i;
    double sqtheta = sqsum_cu(3, rot);
    if (sqtheta != 0)
    {
        double theta, costheta, sintheta, theta_inverse;
        double w[3], w_cross_pt[3], tmp;

        theta = sqrt(sqtheta);
        costheta = cos(theta);
        sintheta = sin(theta);
        theta_inverse = 1.0 / theta;

        for (i = 0; i < 3; i++)
        {
            w[i] = rot[i] * theta_inverse;
        }

        cross_cu(w, pt, w_cross_pt);

        tmp = (w[0] * pt[0] + w[1] * pt[1] + w[2] * pt[2]) *
            (1. - costheta);

        for (i = 0; i < 3; i++)
        {
            rotatedPt[i] = pt[i] * costheta + w_cross_pt[i] * sintheta + w[i] * tmp;
        }
    }
    else
    {
        double rot_cross_pt[3];
        cross_cu(rot, pt, rot_cross_pt);

        for (i = 0; i < 3; i++)
        {
            rotatedPt[i] = pt[i] + rot_cross_pt[i];
        }
    }
}


void __device__ radial_distort_cu(double const* rad_params, double *proj)
{
    double rsq, L;
    rsq = sqsum_cu(2, proj);
    L = 1. + rad_params[0] * rsq + rad_params[1] * rsq * rsq;
    proj[0] = proj[0] * L;
    proj[1] = proj[1] * L;
}


void __device__ project_cu(double const* __restrict cam, double const* __restrict X, double* __restrict proj)
{
    double const* C = &cam[3];
    double Xo[3], Xcam[3];

    Xo[0] = X[0] - C[0];
    Xo[1] = X[1] - C[1];
    Xo[2] = X[2] - C[2];

    rodrigues_rotate_point_cu(&cam[0], Xo, Xcam);

    proj[0] = Xcam[0] / Xcam[2];
    proj[1] = Xcam[1] / Xcam[2];

    radial_distort_cu(&cam[9], proj);

    proj[0] = proj[0] * cam[6] + cam[7];
    proj[1] = proj[1] * cam[6] + cam[8];
}

void __device__   compute_reproj_error_cu(
    double const* __restrict cam,
    double const* __restrict X,
    double const* __restrict w,
    double const* __restrict feat,
    double * __restrict err
)
{
    double proj[2];
    project_cu(cam, X, proj);

    err[0] = (*w)*(proj[0] - feat[0]);
    err[1] = (*w)*(proj[1] - feat[1]);
}

typedef void (*f_ptr)(    double const* __restrict cam,
    double const* __restrict X,
    double const* __restrict w,
    double const* __restrict feat,
    double * __restrict err);

extern void __device__ __enzyme_autodiff(f_ptr,
    int,    double const* cam, double * dcam,
    int , double const* X, double * dX,
    int ,  double const* w, double * wb,
    int ,double const* feat,
    int , double *err,
    double *derr
);
 


void __device__ dcompute_reproj_error_cu( double const* cam,
    double * dcam,
    double const* X,
    double * dX,
    double const* w,
    double * wb,
    double const* feat,
    double *err,
    double *derr
){


     // extern __shared__ double sharedMemory_L[];
    // extern __shared__ double sharedMemory_dL[];
 
 
    __enzyme_autodiff(compute_reproj_error_cu,
            enzyme_dup, cam, dcam,
            enzyme_dup, X, dX,
            enzyme_dup, w, wb,
            enzyme_const, feat,
            enzyme_dupnoneed, err, derr);
 
 
    
 
}


typedef void (*f_ptr)(    double const* __restrict cam,
    double const* __restrict X,
    double const* __restrict w,
    double const* __restrict feat,
    double * __restrict err);

extern void __device__ __enzyme_autodiff(f_ptr,
    int,    double const* cam, double * dcam,
    int , double const* X, double * dX,
    int ,  double const* w, double * wb,
    int ,double const* feat,
    int , double *err,
    double *derr
);

void __device__ compute_zach_weight_error_cu(double const* w, double* err)
{
    *err = 1.0 - (*w)*(*w);
}

typedef void (*f_ptr_weight)(    double const*  w,
    double * err);

extern void __device__ __enzyme_autodiff(f_ptr_weight,
    int,   double const* w, double* dw,
    int , double* err, double* derr
);
void __device__ dcompute_zach_weight_error_cu(double const* w, double* dw, double* err, double* derr) {

    __enzyme_autodiff(compute_zach_weight_error_cu,
            enzyme_dup, w, dw,
            enzyme_dupnoneed, err, derr);
} 



#define BA_NCAMPARAMS 11


 
typedef void(*deriv_reproj_t)(double const*, double *, double const*, double*, double const*, double*, double const*, double *, double *);
__device__ deriv_reproj_t staticTestFunc = dcompute_reproj_error_cu;



void __global__ calculate_reproj_error_jacobian_part_cu__(deriv_reproj_t deriv_reproj,
        int p,
        double *cams,  
        double* cam_gradient_part,
        double *X,
        double* x_gradient_part, 
        double *w,
        double* weight_gradient_part,double *feats,
        int* obs,double *reproj_err_d,double * reproj_err_d_row)
{


    double errb[2];     // stores dY
                        // (i-th element equals to 1.0 for calculating i-th jacobian row)

    double err[2];      // stores fictive result
    //                     // (Tapenade doesn't calculate an original function in reverse mode)

    
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > p)
        {return ;}

    int camIdx =  obs[2 * i + 0];
    int ptIdx =  obs[2 * i + 1];


    // calculate first row
    errb[0] = 1.0;
    errb[1] = 0.0;



    deriv_reproj(
        cams + camIdx * BA_NCAMPARAMS,
        cam_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        X + ptIdx * 3 ,
        x_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        w + i,
        weight_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        feats + i * 2,
        err,
        errb
    );
    // show_res(3,weight_gradient_part);

    // CHECK(hipDeviceSynchronize());
    // CHECK(hipGetLastError());
    // exit(0);
    // fill first row elements
    for (int j = 0; j < BA_NCAMPARAMS + 3 + 1; j++)
    {
        reproj_err_d[2 * j + i * (BA_NCAMPARAMS + 3 + 1) * 2] = reproj_err_d_row[j + i * (BA_NCAMPARAMS + 3 + 1)];
    }

    // calculate second row
    errb[0] = 0.0;
    errb[1] = 1.0;


    deriv_reproj(
        cams + camIdx * BA_NCAMPARAMS,
        cam_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        X + ptIdx * 3 ,
        x_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        w+i,
        weight_gradient_part + i * ( BA_NCAMPARAMS + 3 + 1),
        feats + i * 2,
        err,
        errb
    );
    // show_res(3,weight_gradient_part);


    for (int j = 0; j < BA_NCAMPARAMS + 3 + 1; j++)
    {
        reproj_err_d[2 * j + 1 + i * (BA_NCAMPARAMS + 3 + 1) * 2 ] = reproj_err_d_row[j + i * (BA_NCAMPARAMS + 3 + 1)];
    }


}


__global__ void calculate_weight_error_jacobian_part_cu__(int p,
        double const* w, double* dw, double* err, double* derr){

    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i > p)
        {return ;}
    
    dcompute_zach_weight_error_cu(&w[i],&dw[i],&err[i],&derr[i]);

}

void calculate_weight_error_jacobian_part_cu(int deriv_reproj,int p,
        double const* w, double* dw, double* err, double* derr){

        int dimx = 32;  
        dim3 block(dimx, 1);  
        dim3 grid((p + block.x - 1) / block.x, 1); 
        switch (deriv_reproj)
        {
        case 1:
            calculate_weight_error_jacobian_part_cu__<<<grid,block>>>
            (p,w,dw,err,derr);
        }
}
void  calculate_reproj_error_jacobian_part_cu(int ind,
        int p,
        double *cams,  
        double* cam_gradient_part,
        double *X,
        double* x_gradient_part, 
        double *w,
        double* weight_gradient_part,double *feats,int* obs,
        double *reproj_err_d,double * reproj_err_d_row
        )
        {
                 
        deriv_reproj_t hostFunc;
        //把之前指定过的静态指针复制到host部分
        hipMemcpyFromSymbol(&hostFunc,HIP_SYMBOL(staticTestFunc),sizeof(deriv_reproj_t));

        int dimx = 32;  
        dim3 block(dimx, 1);  
        dim3 grid((p + block.x - 1) / block.x, 1); 
        switch (ind)
        {
        case 1:
            calculate_reproj_error_jacobian_part_cu__<<<grid,block>>>
            (hostFunc,p,cams,cam_gradient_part,X,x_gradient_part,w,weight_gradient_part,feats,
            obs,reproj_err_d,reproj_err_d_row);
            break;
        case 2:
            calculate_reproj_error_jacobian_part_cu__<<<grid,block>>>
            (hostFunc,p,cams,cam_gradient_part,X,x_gradient_part,w,weight_gradient_part,feats,
            obs,reproj_err_d,reproj_err_d_row);
            break;        
        default:
            break;
        }



}

